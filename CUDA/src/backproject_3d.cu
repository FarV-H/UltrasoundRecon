#include "hip/hip_runtime.h"
#include "types.h"
#include "backproject_3d.h"
#include "backproject_3d.cuh"
#include "calc_weight.cuh"
#include "matrix_operator.cuh"

#include <iostream>
#include <fstream>
#include <cstdio>
#include <string>
#include "mex.h"

using namespace std;

void projection_interp(Geometry geo, float* projection, Point_3D* reciecer, float* proj_inter, Point_3D* reciecer_inter, int mul) {
    // 投影数据插值
    const int row_m = mul;
    const int row = (int)(geo.num_channels_x * row_m);
    const int col_m = mul;
    const int col = (int)(geo.num_channels_y * col_m);
    const int srcimg_size = geo.num_channels_x * geo.num_channels_y * sizeof(float);
    const int dstimg_size = row * col * sizeof(float);
    const float x_a = 1.0 / row_m;
    const float y_a = 1.0 / col_m;

    float* D_proj_inter;
    gpuErrchk(hipMalloc((void**)&D_proj_inter, dstimg_size * geo.num_views * geo.num_panel));

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();                             //声明数据类型
    hipArray* cuArray_src;                                                                         //定义CUDA数组
    gpuErrchk(hipMallocArray(&cuArray_src, &channelDesc, geo.num_channels_x, geo.num_channels_y));            //分配大小为col*row的CUDA数组
    tex_src.addressMode[0] = hipAddressModeWrap;                       //寻址方式
    tex_src.addressMode[1] = hipAddressModeWrap;                       //寻址方式 如果是三维数组则设置texRef.addressMode[2]
    tex_src.normalized = false;                                         //是否对纹理坐标归一化
    tex_src.filterMode = hipFilterModeLinear;                          //硬件插值方式：最邻近插值--hipFilterModePoint 双线性插值--hipFilterModeLinear
    gpuErrchk(hipBindTextureToArray(&tex_src, cuArray_src, &channelDesc));        //把CUDA数组绑定到纹理内存

    dim3 Block_resize(16, 16);
    dim3 Grid_resize((row + Block_resize.x - 1) / Block_resize.x, (col + Block_resize.y - 1) / Block_resize.y);

    for (size_t i = 0; i < geo.num_views * geo.num_panel; i++)
    {
        gpuErrchk(hipMemcpyToArray(cuArray_src, 0, 0, projection + i * geo.num_channels, srcimg_size, hipMemcpyHostToDevice));   //把源图像数据拷贝到CUDA数组

        // 调用核函数
        resize_proj_ker << <Grid_resize, Block_resize >> > (row, col, x_a, y_a, D_proj_inter + i * geo.num_channels * row_m * col_m);
        gpuErrchk(hipPeekAtLastError());
    }
    gpuErrchk(hipFreeArray(cuArray_src));
    gpuErrchk(hipUnbindTexture(tex_src));

    // 探测器通道插值
    Point_3D* D_reciecer_inter;
    gpuErrchk(hipMalloc((void**)&D_reciecer_inter, row * col * sizeof(Point_3D) * geo.num_panel));

    //hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();                   //声明数据类型
    hipArray* cuArray_x;                                                                   //定义CUDA数组
    hipArray* cuArray_y;
    hipArray* cuArray_z;
    gpuErrchk(hipMallocArray(&cuArray_x, &channelDesc, geo.num_channels_x, geo.num_channels_y));      //分配大小为col*row的CUDA数组
    gpuErrchk(hipMallocArray(&cuArray_y, &channelDesc, geo.num_channels_x, geo.num_channels_y));
    gpuErrchk(hipMallocArray(&cuArray_z, &channelDesc, geo.num_channels_x, geo.num_channels_y));
    tex_x.addressMode[0] = hipAddressModeWrap;                                             //寻址方式
    tex_y.addressMode[0] = hipAddressModeWrap;
    tex_z.addressMode[0] = hipAddressModeWrap;
    tex_x.addressMode[1] = hipAddressModeWrap;                                             //寻址方式 如果是三维数组则设置texRef.addressMode[2]
    tex_y.addressMode[1] = hipAddressModeWrap;
    tex_z.addressMode[1] = hipAddressModeWrap;
    tex_x.normalized = false;                                                               //是否对纹理坐标归一化
    tex_y.normalized = false;
    tex_z.normalized = false;
    tex_x.filterMode = hipFilterModeLinear;                                                //硬件插值方式：最邻近插值--hipFilterModePoint 双线性插值--hipFilterModeLinear
    tex_y.filterMode = hipFilterModeLinear;
    tex_z.filterMode = hipFilterModeLinear;
    gpuErrchk(hipBindTextureToArray(&tex_x, cuArray_x, &channelDesc));                                //把CUDA数组绑定到纹理内存
    gpuErrchk(hipBindTextureToArray(&tex_y, cuArray_y, &channelDesc));
    gpuErrchk(hipBindTextureToArray(&tex_z, cuArray_z, &channelDesc));

    float* reciecer_x = (float*)malloc(geo.num_channels * geo.num_panel * sizeof(float));
    float* reciecer_y = (float*)malloc(geo.num_channels * geo.num_panel * sizeof(float));
    float* reciecer_z = (float*)malloc(geo.num_channels * geo.num_panel * sizeof(float));
    for (size_t i = 0; i < geo.num_channels * geo.num_panel; i++)
    {
        reciecer_x[i] = reciecer[i].x;
        reciecer_y[i] = reciecer[i].y;
        reciecer_z[i] = reciecer[i].z;
    }
    for (size_t i = 0; i < geo.num_panel; i++)
    {
        gpuErrchk(hipMemcpyToArray(cuArray_x, 0, 0, reciecer_x + i * geo.num_channels, geo.num_channels * sizeof(float), hipMemcpyHostToDevice));   //把数据拷贝到CUDA数组
        gpuErrchk(hipMemcpyToArray(cuArray_y, 0, 0, reciecer_y + i * geo.num_channels, geo.num_channels * sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpyToArray(cuArray_z, 0, 0, reciecer_z + i * geo.num_channels, geo.num_channels * sizeof(float), hipMemcpyHostToDevice));

        resize_reciecer_ker << <Grid_resize, Block_resize >> > (row, col, x_a, y_a, D_reciecer_inter + i * row * col);
        gpuErrchk(hipPeekAtLastError());
    }
    gpuErrchk(hipFreeArray(cuArray_x));
    gpuErrchk(hipFreeArray(cuArray_y));
    gpuErrchk(hipFreeArray(cuArray_z));
    gpuErrchk(hipUnbindTexture(tex_x));
    gpuErrchk(hipUnbindTexture(tex_y));
    gpuErrchk(hipUnbindTexture(tex_z));
    free(reciecer_x);
    free(reciecer_y);
    free(reciecer_z);

    gpuErrchk(hipMemcpy(reciecer_inter, D_reciecer_inter, geo.num_channels * row_m * col_m * geo.num_panel * sizeof(Point_3D), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(proj_inter, D_proj_inter, geo.num_views * geo.num_channels * row_m * col_m * geo.num_panel * sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(D_reciecer_inter));
    gpuErrchk(hipFree(D_proj_inter));

}

float* Backproject3D(Point_3D* source,
    Point_3D* reciecer,
    float* projection,
    float* data,
    Geometry geo) {

    Point_3D* D_source;
    Point_3D* D_reciecer;
    float* D_data;

    gpuErrchk(hipMalloc((void**)&D_source, geo.num_views * geo.num_panel * sizeof(Point_3D)));
    gpuErrchk(hipMalloc((void**)&D_reciecer, geo.num_channels * geo.num_panel * sizeof(Point_3D)));
    gpuErrchk(hipMalloc((void**)&D_data, geo.nx * geo.ny * geo.nz * sizeof(float)));

    gpuErrchk(hipMemcpy(D_source, source, geo.num_views * geo.num_panel * sizeof(Point_3D), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(D_reciecer, reciecer, geo.num_channels * geo.num_panel * sizeof(Point_3D), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(D_geo), &geo, sizeof(struct Geometry)));
    gpuErrchk(hipPeekAtLastError());

    int n = geo.interp;
    float* proj_inter = (float*)malloc(geo.num_views * geo.num_channels * n * n * geo.num_panel * sizeof(float));
    Point_3D* reciecer_inter = (Point_3D*)malloc(geo.num_channels * n * n * geo.num_panel * sizeof(Point_3D));
    projection_interp(geo, projection, reciecer, proj_inter, reciecer_inter, n);




    //ofstream ofs;
    //ofs.open("Point_inter.bin", ios::out | ios::binary);
    //ofs.write((const char*)test, geo.num_panel * dstimg_size);
    //ofs.close();

    //float* test = (float*)malloc(dstimg_size * geo.num_views * geo.num_panel);
    //gpuErrchk(hipMemcpy(test, D_proj_inter, dstimg_size * geo.num_views * geo.num_panel, hipMemcpyDeviceToHost));
    //ofs.open("proj_interp.bin", ios::out | ios::binary);
    //ofs.write((const char*)test, dstimg_size * geo.num_views * geo.num_panel);
    //ofs.close();

    float* weight = (float*)malloc(geo.nx * geo.ny * geo.nz * sizeof(float));
    float* D_weight;
    float* D_weight_proj;
    float* D_weight_sum;
    gpuErrchk(hipMalloc((void**)&D_weight, geo.nx * geo.ny * geo.nz * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&D_weight_proj, geo.nx * geo.ny * geo.nz * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&D_weight_sum, geo.nx * geo.ny * geo.nz * sizeof(float)));
    gpuErrchk(hipMemset(D_weight_sum, 0, geo.nx * geo.ny * geo.nz * sizeof(float)));

    dim3 block(32, 16, 1);
    dim3 grid((geo.nx + block.x - 1) / block.x, (geo.ny + block.y - 1) / block.y, (geo.nz + block.z - 1) / block.z);

    unsigned long index = 0;
    float sum = 0;
    for (size_t k = 0; k < geo.num_panel; k++)
    {
        for (size_t i = 0; i < geo.num_views; i++)
        {
            for (size_t j = 0; j < geo.num_channels * n * n; j++) {

                //mexPrintf("(%d, %d, %d)\n", i, j, k);
                index = k * geo.num_views * geo.num_channels * n * n + i * geo.num_channels * n * n + j;

                memset(weight, 0, geo.nx * geo.ny * geo.nz * sizeof(float));
                gpuErrchk(hipMemset(D_weight, 0, geo.nx * geo.ny * geo.nz * sizeof(float)));

                CalcWeight_kernel << <grid, block >> > (D_weight, source[i + k * geo.num_views], reciecer_inter[j + k * geo.num_channels * n * n]);
                gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipDeviceSynchronize());

                Add_Matrix_kernel << <grid, block >> > (D_weight_sum, D_weight, D_weight_sum, geo.nx, geo.ny, geo.nz);
                gpuErrchk(hipPeekAtLastError());

                gpuErrchk(hipMemcpy(D_weight_proj, D_weight, geo.nx * geo.ny * geo.nz * sizeof(float), hipMemcpyDeviceToDevice));
                MatrixSum << <ceil((double)(geo.nx * geo.ny * geo.nz) / THREAD_LENGTH), THREAD_LENGTH >> > (D_weight, geo.nx * geo.ny * geo.nz);
                //gpuErrchk(hipPeekAtLastError());
                gpuErrchk(hipMemcpy(weight, D_weight, geo.nx * geo.ny * geo.nz * sizeof(float), hipMemcpyDeviceToHost));
                sum = 0;
                for (int l = 0; l < ceil((double)(geo.nx * geo.ny * geo.nz) / THREAD_LENGTH); ++l) {
                    sum += weight[l * THREAD_LENGTH];    //对每个块内部分和求和
                }

                Mul_Matrix_kernel << <grid, block >> > (D_weight_proj, D_weight_proj, proj_inter[index], geo.nx, geo.ny, geo.nz);
                gpuErrchk(hipPeekAtLastError());

                Data_Sum_kernel << <grid, block >> > (D_data, D_weight_proj, sum, geo.nx, geo.ny, geo.nz);
                gpuErrchk(hipPeekAtLastError());
                
                //gpuErrchk(hipMemcpy(data, D_data, geo.nx * geo.ny * geo.nz * sizeof(float), hipMemcpyDeviceToHost))
                //ofstream ofs;
                //ofs.open("test_interp.bin", ios::out | ios::binary);
                //ofs.write((const char*)data, geo.nx* geo.ny* geo.nz * sizeof(float));
                //ofs.close();
            }
        }
    }
    //mexPrintf("AA\n");
    
    //gpuErrchk(hipMemcpy(data, D_weight_sum, geo.nx * geo.ny * geo.nz * sizeof(float), hipMemcpyDeviceToHost))

    //ofstream ofs;
    //ofs.open("test_interp.bin", ios::out | ios::binary);
    //ofs.write((const char*)data, geo.nx * geo.ny * geo.nz * sizeof(float));
    //ofs.close();

    // 反投影数据除以总走时
    Div_Matrix_kernel << <grid, block >> > (D_data, D_weight_sum, D_data, geo.nx, geo.ny, geo.nz);
    gpuErrchk(hipPeekAtLastError());
    
    gpuErrchk(hipFree(D_weight_proj));
    gpuErrchk(hipFree(D_source));
    gpuErrchk(hipFree(D_reciecer));
    free(weight);
    //mexPrintf("BB\n");
    
    gpuErrchk(hipMemcpy(data, D_data, geo.nx * geo.ny* geo.nz * sizeof(float), hipMemcpyDeviceToHost))

    //ofstream ofs;
    //ofs.open("test_interp.bin", ios::out | ios::binary);
    //ofs.write((const char*)data, geo.nx* geo.ny* geo.nz * sizeof(float));
    //ofs.close();
    //mexPrintf("BB\n");

    gpuErrchk(hipFree(D_weight));
    gpuErrchk(hipFree(D_data));

    return data;
}